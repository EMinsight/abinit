//#if defined HAVE_CONFIG_H
#include "config.h"
//#endif

#include "abi_common.h"
#include "stdio.h"
#if defined HAVE_GPU_CUDA
//#include "abi_gpu_header.h"
#include "cuda_header.h"
#include "cuda_api_error_check.h"
#endif


extern "C" void gpu_planpp_free(void **plan_pp) {
  hipfftHandle plan = * ((hipfftHandle *) (*plan_pp));
  CHECK_CUDA_ERROR(hipfftDestroy(plan));
}


extern "C" void devptr_free(void *dev_ptr) {
  CHECK_CUDA_ERROR(hipFree(dev_ptr));
}


extern "C" void xgpu_fftbox_c2c_ip(int *f_dims, int *f_embed, int ndat, int isign, int kind,
                                   void *h_ff, void **plan_pp, void *d_ff) {
                                   //hipfftComplex *h_ff, hipfftHandle *plan_pp, hipfftComplex *d_ff) {

  const int RANK = 3, stride = 1;
  size_t nbytes;
  int c_dims[RANK], c_embed[RANK], dist, direction;
  c_dims[0] = f_dims[2]; c_dims[1] = f_dims[1]; c_dims[2] = f_dims[0];
  c_embed[0] = f_embed[2]; c_embed[1] = f_embed[1]; c_embed[2] = f_embed[0];
  dist = f_embed[0] * f_embed[1] * f_embed[2];

#if defined HAVE_GPU_CUDA
  hipfftType type;
  hipfftHandle plan;
  switch (kind) {
  case 4:
    type = HIPFFT_C2C;
    nbytes = dist * ndat * sizeof(hipfftComplex);
    break;
  case 8:
    type = HIPFFT_Z2Z;
    nbytes = dist * ndat * sizeof(hipfftDoubleComplex);
    break;
  default:
    printf("Invalid kind: %d\n", kind);
    abi_cabort();
  }

  switch (isign) {
  case 1:
    direction = HIPFFT_BACKWARD;
    break;
  case -1:
    direction = HIPFFT_FORWARD;
    break;
  default:
    printf("Invalid isign: %d\n", isign);
    abi_cabort();
  }

  if (d_ff == NULL) {
    printf("Calling hipMalloc");
    CHECK_CUDA_ERROR(hipMalloc((void**) &d_ff, nbytes));
  }

  CHECK_CUDA_ERROR(hipMemcpy(d_ff, h_ff, nbytes, hipMemcpyHostToDevice));

  /* Create a 3D FFT plan.
  hipfftResult = hipfftPlanMany(hipfftHandle *plan, int rank, int *c_dims,
                              int *inembed, int istride, int idist,
                              int *onembed, int ostride, int odist,
                              hipfftType type, int batch);
  */

  if (*plan_pp == NULL) {
    printf("Building plan");
    CHECK_CUDA_ERROR(hipfftPlanMany(&plan, RANK, c_dims, c_embed, stride, dist, c_embed, stride, dist, type, ndat));
	  //*plan_pp = (void **) &plan;
  }
  else {
    printf("Reusing plan");
	  plan = * ((hipfftHandle *) (*plan_pp));
	}

  /* Transform the signal in place. */
  if (type == HIPFFT_C2C) {
		CHECK_CUDA_ERROR(hipfftExecC2C(plan, (hipfftComplex *) d_ff, (hipfftComplex *) d_ff, direction));
	}
  if (type == HIPFFT_Z2Z) {
		CHECK_CUDA_ERROR(hipfftExecZ2Z(plan, (hipfftDoubleComplex *) d_ff, (hipfftDoubleComplex *) d_ff, direction));
	}

  CHECK_CUDA_ERROR(hipDeviceSynchronize());
  CHECK_CUDA_ERROR(hipMemcpy(h_ff, d_ff, nbytes, hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipFree(d_ff));
  CHECK_CUDA_ERROR(hipfftDestroy(plan));
#endif
}
